#include "hip/hip_runtime.h"

// Source code for the CUDA kernel 

extern "C"{
    #include "sparse_mult.cuh"
}

#include <stdio.h>


#ifndef DEBUG
// #define DEBUG
#endif


#ifndef DEBUG_GUDA_SOME
// #define DEBUG_GUDA_SOME
#endif





extern __global__ void whoami(void){

    int block_id = 
                    blockIdx.x + 
                    blockIdx.y * gridDim.x + 
                    blockIdx.z * gridDim.x * gridDim.y ;

    int block_offset = 
                    block_id * blockDim.x * blockDim.y * blockDim.z ;
    
    
    int thread_offset = 
                    threadIdx.x + 
                    threadIdx.y * blockDim.x + 
                    threadIdx.z * blockDim.x * blockDim.y ;

    int id = block_offset + thread_offset ; 

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d \n" , 
        id , 
        blockIdx.x  , blockIdx.y  , blockIdx.z  , block_offset , 
        threadIdx.x , threadIdx.y , threadIdx.z , thread_offset ) ;
    
}


extern __global__ void sparse_mult( int N , int nz  ,  int* pointers , int* nz_elements , int* results ){ 

    // Find thread id 
    int id = blockIdx.x * blockDim.x + threadIdx.x ; 

    
    
    // #ifdef DEBUG_GUDA_SOME
    // printf("Hi i am the thread number %4d   am i greater than N %d!!! \n", id ,  id >= N ) ;
    
    // #endif


    if ( id >= N  ){
        #ifdef DEBUG_GUDA_SOME
        printf("Hi i am the thread number %4d and i will not do anything !!! \n", id  ) ;
        #endif
        return ; 
    }


    // Print thread id 
    #ifdef DEBUG
        printf("%4d | Block(%d %d %d) | Thread(%d %d %d) | Start and stop at: pointers[%3d]= %3d  , pointers[%3d]= %3d \n" , 
            id  , 
            blockIdx.x  , blockIdx.y  , blockIdx.z   , 
            threadIdx.x , threadIdx.y , threadIdx.z  , 
            id , pointers[id] , id+1, pointers[id+1]) ;
    #endif


    results[id] = 0 ; 
    for( int other_point_pointer = pointers[id] ; other_point_pointer < pointers[id+1] ; other_point_pointer++){
        // printf("%04d | Block(%d %d %d) | Thread(%d %d %d) | pointers[%2d] = %2d \n" , 
        // id  , 
        // blockIdx.x  , blockIdx.y  , blockIdx.z   , 
        // threadIdx.x , threadIdx.y , threadIdx.z  , 
        // other_point_pointer , nz_elements[other_point_pointer]) ;
        
        int other_point_id = nz_elements[other_point_pointer] ; 

        if( other_point_id < id ){
            continue ; 
        }

        if (  nz_elements[pointers[id]] > nz_elements[pointers[other_point_id+1]-1]   || nz_elements[pointers[id+1]-1] < nz_elements[pointers[other_point_id]]  ){
            // This means there is not overlap between the two lists
            #ifdef DEBUG
            printf("The lists %3d and %3d  do not have any overlap\n" , id , other_point_id );
            #endif
            continue ; 
        }
        // We need to find how many common points there are between "id" and "other_point_id"


        // 'pointers' to element
        int mine_start = pointers[id] ;
        int mine_end = pointers[id+1] ;
        int other_start = pointers[other_point_id] ;
        int other_end = pointers[other_point_id+1] ;

        
        #ifdef DEBUG
        int count = 0 ; 
        #endif

        while( mine_start < mine_end && other_start < other_end ){

            
            #ifdef DEBUG
            // printf("Common check between lists: %3d and %3d , now checking elements %3d with %3d -------\n" , id , other_point_id ,  nz_elements[mine_start] , nz_elements[other_start]  );
            #endif

            if ( nz_elements[mine_start] < nz_elements[other_start] ){
                mine_start++;
            }
            else if ( nz_elements[mine_start] > nz_elements[other_start]  ){
                other_start++;
            }
            else{
                
                results[id]++;
                other_start++;
                mine_start++;

                #ifdef DEBUG
                count++; 
                #endif
            }

        }

        #ifdef DEBUG 
        printf("Common elements found between %3d and %3d are %3d many \n" , id , other_point_id , count);
        #endif


    }


    #ifdef DEBUG
    printf("%04d | Block(%d %d %d) | Thread(%d %d %d) | End | results[%3d] = %3d  \n" , 
        id  , 
        blockIdx.x  , blockIdx.y  , blockIdx.z   , 
        threadIdx.x , threadIdx.y , threadIdx.z  , 
        id , results[id]) ;
    #endif
    
}


extern __global__ void find_d2_hat( int N , int nz  ,  int* pointers , int* nz_elements ,int* p1 ,  int* results ){ 

    // Find thread id 
    int id = blockIdx.x * blockDim.x + threadIdx.x ; 


    if ( id >= N  ){
        #ifdef DEBUG_GUDA_SOME
            printf("Hi i am the thread number %4d and i will not do anything !!! \n", id  ) ;
        #endif
        return ; 
    }


    // Print thread id 
    #ifdef DEBUG
        printf("%4d | Block(%d %d %d) | Thread(%d %d %d) | Start and stop at: pointers[%3d]= %3d  , pointers[%3d]= %3d \n" , 
            id  , 
            blockIdx.x  , blockIdx.y  , blockIdx.z   , 
            threadIdx.x , threadIdx.y , threadIdx.z  , 
            id , pointers[id] , id+1, pointers[id+1]) ;
    #endif

    // Perform  A*p1-p1 
    results[id] = - p1[id] ; 
    for( int other_point_pointer = pointers[id] ; other_point_pointer < pointers[id+1] ; other_point_pointer++){
        // printf("%04d | Block(%d %d %d) | Thread(%d %d %d) | pointers[%2d] = %2d \n" , 
        // id  , 
        // blockIdx.x  , blockIdx.y  , blockIdx.z   , 
        // threadIdx.x , threadIdx.y , threadIdx.z  , 
        // other_point_pointer , nz_elements[other_point_pointer]) ;
        
        int other_point_id = nz_elements[other_point_pointer] ; 

        results[id] += p1[other_point_id] ; 

    }


    #ifdef DEBUG
        printf("%04d | Block(%d %d %d) | Thread(%d %d %d) | End | results[%3d] = %3d  \n" , 
            id  , 
            blockIdx.x  , blockIdx.y  , blockIdx.z   , 
            threadIdx.x , threadIdx.y , threadIdx.z  , 
            id , results[id]) ;
    #endif
    
}


__global__ void find_d3_hat( int N , int* p1  , int* d3_hat ){ 

    // Find thread id 
    int id = blockIdx.x * blockDim.x + threadIdx.x ; 

    if ( id >= N  ){
        #ifdef DEBUG_GUDA_SOME
        printf("Hi i am the thread number %4d and i will not do anything inside find_d3_hat  !!! \n", id  ) ;
        #endif
        return ; 
    }

    d3_hat[id] =  p1[id] * (p1[id]-1) / 2  ; 

    #ifdef DEBUG
    printf("%04d | Block(%d %d %d) | Thread(%d %d %d) | End | d3_hat[%3d] = %3d  \n" , 
        id  , 
        blockIdx.x  , blockIdx.y  , blockIdx.z   , 
        threadIdx.x , threadIdx.y , threadIdx.z  , 
        id , d3_hat[id]) ;
    #endif
    
}



// static const int wholeArraySize = 100000000;
static const int blockSize = 1024;
static const int gridSize = 24; //this number is hardware-dependent; usually #SM*2 is a good number.

__global__ void sumCommMultiBlock(const int *gArr, int arraySize, int *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}



__host__ int sumArray(int* arr , int wholeArraySize ) {
    int* dev_arr;
    hipMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    hipMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), hipMemcpyHostToDevice);

    int out;
    int* dev_out;
    hipMalloc((void**)&dev_out, sizeof(int)*gridSize);
    
    sumCommMultiBlock<<<gridSize, blockSize>>>(dev_arr, wholeArraySize, dev_out);
    //dev_out now holds the partial result
    sumCommMultiBlock<<<1, blockSize>>>(dev_out, gridSize, dev_out);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();
    
    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_arr);
    hipFree(dev_out);
    return out;
}


__host__ int sumArray_already_in_GPU(int* dev_arr , int wholeArraySize ) {
    // int* dev_arr;
    // hipMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    // hipMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), hipMemcpyHostToDevice);

    int out;
    int* dev_out;
    hipMalloc((void**)&dev_out, sizeof(int)*gridSize);
    
    sumCommMultiBlock<<<gridSize, blockSize>>>(dev_arr, wholeArraySize, dev_out);
    //dev_out now holds the partial result
    sumCommMultiBlock<<<1, blockSize>>>(dev_out, gridSize, dev_out);
    //dev_out[0] now holds the final result
    hipDeviceSynchronize();
    
    hipMemcpy(&out, dev_out, sizeof(int), hipMemcpyDeviceToHost);
    // hipFree(dev_arr);
    hipFree(dev_out);
    return out;
}
