
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>


#include "sparse_mult.cuh"


extern "C" {
    #include "load_stuff.h"
    #include "csx.h" // Credits to JohnStef :) 
}

#ifndef DEBUG_PRINT_ONLY_TIME
// #define DEBUG_PRINT_ONLY_TIME
#endif


#ifndef DEBUG_TIME_CPP
// #define DEBUG_TIME_CPP
#endif


// nvcc    -O3  main.cu    load_stuff.c  sparse_mult.cu  mmio.c csx.c -o CUDA_nick.exe

int main ( int argc, char* argv[] ){


    // Choose input 
    
    char file_relative_path[100] = "../Data" ; 

    
    if (argc > 2) {
        printf("Usage: %s <integer>\n", argv[0]);
        // return 1;
    }

    char* file_to_load;

    int option_chosen = - 1 ; 

    if ( argc == 2  ){

        option_chosen = atoi(argv[1]);
        printf("The option_chosen is: %d\n", option_chosen);

        file_to_load = decide_file_to_load( option_chosen , file_relative_path, -1 ); 
    }
    else{
        file_to_load = select_file_to_load(file_relative_path ); 
    }
    

    #ifndef DEBUG_PRINT_ONLY_TIME
        printf("Start loading file: %s \n\n" , file_to_load ) ;
    #endif


    

    // Load data 
    
    int* p1 ;
    csx my_sparse =  csc_from_file(file_to_load, &p1);
    

    int* compressed_columes     = my_sparse->com ; 
    int* full_rows              = my_sparse->unc ;
    int nz          = my_sparse->e ;
    int matrix_dim  = my_sparse->v ; 

    
    // csx_print(my_sparse);

    #ifndef DEBUG_PRINT_ONLY_TIME
        printf("Loading Completed!\n"  ) ;
        printf("Matrix N = %d    |  nz = %d \n" , matrix_dim , nz ) ; 
    #endif

    
    
    
    // Send data to GPU 

        // Mesure time
        
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start);
        

        #ifdef DEBUG_TIME_CPP
            auto cpp_start = std::chrono::high_resolution_clock::now();
        #endif


    int* d_nz;
    int* d_pointers;
    int* d_result_vector ; 
    int* d_p1 ;    
    
    hipMalloc( (void**)&d_nz            ,  nz              * sizeof(int) ) ; 
    hipMalloc( (void**)&d_pointers      , (matrix_dim+1)   * sizeof(int) ) ; 
    hipMalloc( (void**)&d_result_vector , (matrix_dim)     * sizeof(int) ) ; 


    hipMalloc( (void**)&d_p1            ,  matrix_dim * sizeof(int) ) ;     
    hipMemcpy( (void*)d_p1         , p1                    ,  matrix_dim       * sizeof(int)   , hipMemcpyHostToDevice );
    
    
    hipMemcpy( (void*)d_nz         , full_rows             ,       nz          * sizeof(int)   , hipMemcpyHostToDevice );
    hipMemcpy( (void*)d_pointers   , compressed_columes    , (matrix_dim+1)    * sizeof(int)   , hipMemcpyHostToDevice );

    
    // Make the calcualation in GPU for C3


    int threads_per_block = 1024 ; // This can not be greater than 1024 !!!
    int number_of_blocks = (matrix_dim/threads_per_block+1) ;

    #ifndef DEBUG_PRINT_ONLY_TIME
    //printf("Created %d blocks with %d threads each, total of %d threads\n" , number_of_blocks , threads_per_block , number_of_blocks * threads_per_block );
    #endif  

    sparse_mult<<<number_of_blocks,threads_per_block>>>(matrix_dim , nz ,d_pointers, d_nz  , d_result_vector ) ; 






    // Find sigma4 
    
    // Take d_result_vector  (it had C3) 
    int sigma4_from_GPU = sumArray_already_in_GPU(d_result_vector , matrix_dim ) ;
    // release d_result_vector  = C3



    // Take d_result_vector  (it had C3) 
    find_d3_hat<<<number_of_blocks,threads_per_block>>>( matrix_dim , d_p1  , d_result_vector );

    int sigma3_hat_from_GPU = sumArray_already_in_GPU(d_result_vector , matrix_dim ) ;  // Sum d3_hat to find sigma3 
    // release d_result_vector = d3_hat


    // Take d_result_vector  (it had d3_hat) 
    find_d2_hat<<<number_of_blocks,threads_per_block>>>( matrix_dim,  nz  ,  d_pointers, d_nz  , d_p1 ,  d_result_vector );
    
    int sigma2_hat_from_GPU = sumArray_already_in_GPU(d_result_vector , matrix_dim ) ;  // Sum d2_hat to find sigma2
    // release d_result_vector = d2_hat


        
    hipDeviceSynchronize() ; // Probably not needed for the time

        // Time taken 

        hipEventRecord(end);
        hipEventSynchronize(start);
        hipEventSynchronize(end);

        float time_in_ms = 0;
        hipEventElapsedTime(&time_in_ms, start, end);

    
        #ifndef DEBUG_PRINT_ONLY_TIME
            // Normal print
            printf("Time %f ms  \n" , time_in_ms );
        #endif
        
        #ifdef DEBUG_PRINT_ONLY_TIME
            printf("Time %f ms    <---------- CUDA  for file: %s\n" , time_in_ms , file_to_load);
        #endif


        #ifdef DEBUG_TIME_CPP
            auto cpp_end = std::chrono::high_resolution_clock::now();
            double cpp_time = std::chrono::duration_cast<std::chrono::milliseconds>(cpp_end-cpp_start).count();
            printf("Time %f ms    <---------- CPP   for file: %s \n" , cpp_time , file_to_load);
        #endif


        




    // Print resutls σ1 σ2 σ3 σ4

    int sigma[5] ;
    sigma[0] =  matrix_dim ; 
    sigma[1] =  nz ; 
    sigma[2] =  sigma2_hat_from_GPU - 2 * sigma4_from_GPU  ; 
    sigma[3] =  sigma3_hat_from_GPU -     sigma4_from_GPU  ; 
    sigma[4] =  sigma4_from_GPU ; 

    #ifdef DEBUG_PRINT_ONLY_TIME
        check_results( sigma , option_chosen ) ; 
    #endif

    #ifndef DEBUG_PRINT_ONLY_TIME
        printf("\nResults\n" ) ;
        printf("sigma0: %d \n" , sigma[0] ) ; 
        printf("sigma1: %d \n" , sigma[1] ) ; 
        printf("sigma2: %d \n" , sigma[2] ) ; 
        printf("sigma3: %d \n" , sigma[3] ) ; 
        printf("sigma4: %d \n" , sigma[4] ) ; 
    #endif  
    

    // Clear memory both in CPU and GPU 
    hipFree((void*)d_nz);
    hipFree((void*)d_pointers);
    hipFree((void*)d_result_vector);
    hipFree((void*)d_p1);
    
    free(file_to_load);
    free(p1); 
    
    csx_free( my_sparse ) ; 


    return 0 ; 

}